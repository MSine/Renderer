#include "hip/hip_runtime.h"
#include "renderer.hpp"

__device__ Mesh mesh;
__device__ Vec3D camPos(5000, 2500, 9000);

__global__ sf::Color rayCast(Vec3D rayDir) {
    uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
    Vec3D intersection;
    Ray ray(camPos.pos, x, y, -FDP);

    bool rayActive;

    do {
        rayActive = false;
        for (auto trig: mesh.triangles) {
            if (trig->rayIntersectsTriangle(ray, intersection)) {
                //TODO  make surface brightness effect the color, arctan(energy*brightness) maybe?
                if (trig->surface.brightness > 0) {
                    ray.distTrav += ray.rayStart.distance(intersection);
                    return ray.adjustedColor();
                }
                else if (ray.reflect(intersection, *trig)) {
                    rayActive = true;
                    continue;
                }
            }
        }
    } while (rayActive);

    return sf::Color::Black;
}